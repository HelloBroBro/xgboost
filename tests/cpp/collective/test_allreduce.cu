
#include <hip/hip_runtime.h>
/**
 * Copyright 2023-2024, XGBoost Contributors
 */
#if defined(XGBOOST_USE_NCCL)
#include <gtest/gtest.h>
#include <thrust/host_vector.h>  // for host_vector

#include "../../../src/common/cuda_rt_utils.h"     // for AllVisibleGPUs
#include "../../../src/common/device_helpers.cuh"  // for ToSpan,  device_vector
#include "../../../src/common/type.h"              // for EraseType
#include "test_worker.cuh"                         // for NCCLWorkerForTest
#include "test_worker.h"                           // for WorkerForTest, TestDistributed

namespace xgboost::collective {
namespace {
class MGPUAllreduceTest : public SocketTest {};

class Worker : public NCCLWorkerForTest {
 public:
  using NCCLWorkerForTest::NCCLWorkerForTest;

  void BitOr() {
    dh::device_vector<std::uint32_t> data(comm_.World(), 0);
    data[comm_.Rank()] = ~std::uint32_t{0};
    auto rc = nccl_coll_->Allreduce(*nccl_comm_, common::EraseType(dh::ToSpan(data)),
                                    ArrayInterfaceHandler::kU4, Op::kBitwiseOR);
    SafeColl(rc);
    thrust::host_vector<std::uint32_t> h_data(data.size());
    thrust::copy(data.cbegin(), data.cend(), h_data.begin());
    for (auto v : h_data) {
      ASSERT_EQ(v, ~std::uint32_t{0});
    }
  }

  void Acc() {
    dh::device_vector<double> data(314, 1.5);
    auto rc = nccl_coll_->Allreduce(*nccl_comm_, common::EraseType(dh::ToSpan(data)),
                                    ArrayInterfaceHandler::kF8, Op::kSum);
    SafeColl(rc);
    for (std::size_t i = 0; i < data.size(); ++i) {
      auto v = data[i];
      ASSERT_EQ(v, 1.5 * static_cast<double>(comm_.World())) << i;
    }
  }
};
}  // namespace

TEST_F(MGPUAllreduceTest, BitOr) {
  auto n_workers = curt::AllVisibleGPUs();
  TestDistributed(n_workers, [=](std::string host, std::int32_t port, std::chrono::seconds timeout,
                                 std::int32_t r) {
    Worker w{host, port, timeout, n_workers, r};
    w.Setup();
    w.BitOr();
  });
}

TEST_F(MGPUAllreduceTest, Sum) {
  auto n_workers = curt::AllVisibleGPUs();
  TestDistributed(n_workers, [=](std::string host, std::int32_t port, std::chrono::seconds timeout,
                                 std::int32_t r) {
    Worker w{host, port, timeout, n_workers, r};
    w.Setup();
    w.Acc();
  });
}
}  // namespace xgboost::collective
#endif  // defined(XGBOOST_USE_NCCL)
